#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include "get_walltime.h"
using namespace std;

const unsigned int NG = 2;
const unsigned int BLOCK_DIM_X = 256;

__constant__ float c_a, c_b, c_c, c_dt, c_dx;

/********************************************************************************
  Error checking function for CUDA
 *******************************************************************************/
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
//https://github.com/parallel-forall/code-samples/blob/master/series/cuda-cpp/finite-difference/finite-difference.cu
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

/********************************************************************************
  Do one diffusion step, on the host in host memory
 *******************************************************************************/
void host_diffusion(float* u, float *u_new, const unsigned int n,
     const float dx, const float dt){

  //First, do the diffusion step on the interior points
  for(int i = NG; i < n-NG;i++){
    u_new[i] = u[i] + dt/(dx*dx) *(
                    - 1./12.f* u[i-2]
                    + 4./3.f * u[i-1]
                    - 5./2.f * u[i]
                    + 4./3.f * u[i+1]
                    - 1./12.f* u[i+2]);
  }

  //Apply the dirichlet boundary conditions
  u_new[0] = -u_new[NG+1];
  u_new[1] = -u_new[NG];

  u_new[n-NG]   = -u_new[n-NG-1];
  u_new[n-NG+1] = -u_new[n-NG-2];
}
/********************************************************************************
  Do one diffusion step, with CUDA
 *******************************************************************************/
__global__
void cuda_diffusion(
    float* u,
    float *u_new,
    const unsigned int n
  ){

  int i = threadIdx.x;

  if (i >= NG && i < n - NG) {
    //Do the diffusion
     u_new[i] = u[i] + c_dt/(c_dx*c_dx) *(
                    - c_a * u[i-2]
                    + c_b * u[i-1]
                    - c_c * u[i]
                    + c_b * u[i+1]
                    - c_a * u[i+2]);
  //Apply the dirichlet boundary conditions
  //HINT: Think about which threads will have the data for the boundaries
  } else if (i == 0) {
    u_new[0] = -u_new[NG+1];
  } else if (i == 1) {
    u_new[1] = -u_new[NG];
  } else if (i == n-NG) {
    u_new[n-NG]   = -u_new[n-NG-1];
  } else if (i == n-NG+1) {
    u_new[n-NG+1] = -u_new[n-NG-2];
  }
}

/********************************************************************************
  Do one diffusion step, with CUDA, with shared memory
 *******************************************************************************/
__global__
void shared_diffusion(float* u, float *u_new, const unsigned int n){

  //Allocate the shared memory
  //FIXME

  //Fill shared memory with the data needed from global memory
  //HINT:
  //What data does each block need from global memory?
  //When do the threads in the block need to sync?
  //FIXME

  //Do the diffusion
  //FIXME

  //Apply the dirichlet boundary conditions
  //HINT: Think about which threads will have the data for the boundaries
  //FIXME
}

/********************************************************************************
  Dump u to a file
 *******************************************************************************/
void outputToFile(string filename, float* u, unsigned int n){

  ofstream file;
  file.open(filename.c_str());
  file.precision(8);
  file << std::scientific;
  for(int i =0; i < n;i++){
    file<<u[i]<<endl;
  }
  file.close();
};

/********************************************************************************
  main
 *******************************************************************************/
int main(int argc, char** argv){

  //Number of steps to iterate
  // const unsigned int n_steps = 10;
  const unsigned int n_steps = 100;
  // const unsigned int n_steps = 1000000;

  //Whether and how ow often to dump data
  const bool outputData = true;
  const unsigned int outputPeriod = n_steps/10;

  //Size of u
  //const unsigned int n = (1<<11) +2*NG;
  const unsigned int n = (1<<16) +2*NG;

  //Block and grid dimensions
  const unsigned int blockDim = BLOCK_DIM_X;
  const unsigned int gridDim = (n-2*NG)/blockDim;

  //Physical dimensions of the domain
  const float L = 2*M_PI;
  const float dx = L/(n-2*NG-1);
  const float dt = 0.25*dx*dx;

  //Create constants for 6th order centered 2nd derivative
  float const_a = 1.f/12.f * dt/(dx*dx);
  float const_b = 4.f/3.f  * dt/(dx*dx);
  float const_c = 5.f/2.f  * dt/(dx*dx);

  //Copy these the cuda constant memory
  checkCuda(
    hipMemcpyToSymbol(HIP_SYMBOL(c_a), &const_a, sizeof(float), 0, hipMemcpyHostToDevice)
  );
  checkCuda(
    hipMemcpyToSymbol(HIP_SYMBOL(c_b), &const_b, sizeof(float), 0, hipMemcpyHostToDevice)
  );
  checkCuda(
    hipMemcpyToSymbol(HIP_SYMBOL(c_c), &const_c, sizeof(float), 0, hipMemcpyHostToDevice)
  );
  checkCuda(
    hipMemcpyToSymbol(HIP_SYMBOL(c_dt), &dt, sizeof(float), 0, hipMemcpyHostToDevice)
  );
  checkCuda(
    hipMemcpyToSymbol(HIP_SYMBOL(c_dx), &dx, sizeof(float), 0, hipMemcpyHostToDevice)
  );

  //iterator, for later
  int i;

  //Create cuda timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

  //Timing variables
  float milliseconds;
  double startTime,endTime;

  //Filename for writing
  char filename[256];

  //Allocate memory for the initial conditions
  float* initial_u = new float[n];

  //Initialize with a periodic sin wave that starts after the left hand
  //boundaries and ends just before the right hand boundaries
  for( i = NG; i < n-NG; i++){
    initial_u[i] = sin( 2*M_PI/L*(i-NG)*dx);
  }
  //Apply the dirichlet boundary conditions
  initial_u[0] = -initial_u[NG+1];
  initial_u[1] = -initial_u[NG];

  initial_u[n-NG]   = -initial_u[n-NG-1];
  initial_u[n-NG+1] = -initial_u[n-NG-2];

/********************************************************************************
  Test the host kernel for diffusion
 *******************************************************************************/

  //Allocate memory in the host's heap
  float* host_u  = new float[n];
  float* host_u2 = new float[n];//buffer used for u_new

  //Initialize the host memory
  for( i = 0; i < n; i++){
    host_u[i] = initial_u[i];
  }

  outputToFile("data/host_uInit.dat",host_u,n);


  get_walltime(&startTime);
  //Perform n_steps of diffusion
  for( i = 0 ; i < n_steps; i++){

    if(outputData && i%outputPeriod == 0){
      sprintf(filename,"data/host_u%08d.dat",i);
      outputToFile(filename,host_u,n);
    }

    host_diffusion(host_u,host_u2,n,dx,dt);

    //Switch the buffer with the original u
    float* tmp = host_u;
    host_u = host_u2;
    host_u2 = tmp;

  }
  get_walltime(&endTime);

  cout<<"Host function took: "<<(endTime-startTime)*1000./n_steps<<"ms per step"<<endl;

  outputToFile("data/host_uFinal.dat",host_u,n);

/********************************************************************************
  Test the cuda kernel for diffusion
 *******************************************************************************/
  //Allocate a copy for the GPU memory in the host's heap
  float* cuda_u  = new float[n];

  //Initialize the cuda memory
  for( i = 0; i < n; i++){
    cuda_u[i] = initial_u[i];
  }
  outputToFile("data/cuda_uInit.dat",cuda_u,n);

  //Allocate memory on the GPU
  float *d_u, *d_u2;
  checkCuda(
    hipMalloc(&d_u, n*sizeof(float))
  );
  checkCuda(
    hipMalloc(&d_u2, n*sizeof(float))
  );

  //Copy cuda_u into d_u
  checkCuda(
    hipMemcpy(d_u, cuda_u, n*sizeof(float), hipMemcpyHostToDevice)
  );


	hipEventRecord(start);//Start timing
  //Perform n_steps of diffusion
  for( i = 0 ; i < n_steps; i++){

    if(outputData && i%outputPeriod == 0){

      sprintf(filename,"data/cuda_u%08d.dat",i);

      //Copy data off the device for writing
      checkCuda(
        hipMemcpy(cuda_u, d_u, n*sizeof(float), hipMemcpyDeviceToHost)
      );
      outputToFile(filename,cuda_u,n);

    }

    //Call the cuda_diffusion kernel
    cuda_diffusion<<<gridDim,blockDim>>>(d_u,d_u2,n);

    //Switch the buffer with the original u
    float *temp = d_u;
    d_u = d_u2;
    d_u2 = temp;

  }
	hipEventRecord(stop);//End timing


  //one last data dump...
  sprintf(filename,"data/cuda_u%08d.dat",i);

  // Copy the memory back for one last data dump
  checkCuda(
    hipMemcpy(cuda_u, d_u, n*sizeof(float), hipMemcpyDeviceToHost)
  );
  outputToFile(filename,cuda_u,n);

  //Get the total time used on the GPU
	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

  cout<<"Cuda Kernel took: "<<milliseconds/n_steps<<"ms per step"<<endl;


/********************************************************************************
  Test the cuda kernel for diffusion with shared memory
 *******************************************************************************/

  //Allocate a copy for the GPU memory in the host's heap
  float* shared_u  = new float[n];

  /*
  //Initialize the cuda memory
  for( i = 0; i < n; i++){
    shared_u[i] = initial_u[i];
  }
  outputToFile("data/shared_uInit.dat",shared_u,n);

  //Copy the initial memory onto the GPU
  //FIXME copy shared_u to d_u



	hipEventRecord(start);//Start timing
  //Perform n_steps of diffusion
  for( i = 0 ; i < n_steps; i++){

    if(outputData && i%outputPeriod == 0){
      //Copy data off the device for writing
      sprintf(filename,"data/shared_u%08d.dat",i);
      //FIXME

      outputToFile(filename,shared_u,n);
    }

    //Call the shared_diffusion kernel
    //FIXME

    //Switch the buffer with the original u
    //FIXME

  }
	hipEventRecord(stop);//End timing


  //Copy the memory back for one last data dump
  sprintf(filename,"data/shared_u%08d.dat",i);
  //FIXME


  //Get the total time used on the GPU
	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

  cout<<"Shared Memory Kernel took: "<<milliseconds/n_steps<<"ms per step"<<endl;
  */

/********************************************************************************
  Test the cuda kernel for diffusion, with excessive memcpys
 *******************************************************************************/

  /*
  //Initialize the cuda memory
  for( i = 0; i < n; i++){
    shared_u[i] = initial_u[i];
  }

	hipEventRecord(start);//Start timing
  //Perform n_steps of diffusion
  for( i = 0 ; i < n_steps; i++){

    //Copy the data from host to device
    //FIXME copy shared_u to d_u

    //Call the shared_diffusion kernel
    //FIXME

    //Copy the data from host to device
    //FIXME copy d_u2 to cuda_u


  }
	hipEventRecord(stop);//End timing



  //Get the total time used on the GPU
	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

  cout<<"Excessive hipMemcpy took: "<<milliseconds/n_steps<<"ms per step"<<endl;
  */

  //Clean up the data
  delete[] initial_u;
  delete[] host_u;
  delete[] host_u2;

  delete[] cuda_u;
  delete[] shared_u;

  //free d_u and d_u2
  hipFree(d_u);
  hipFree(d_u2);
}
